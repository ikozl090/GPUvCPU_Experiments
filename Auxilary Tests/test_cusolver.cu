#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

int main() {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;

    const int n = 3;  // Size of the matrix
    const int lda = n; // Leading dimension of A
    float A[lda*n] = {  // The matrix A
        1.0, 2.0, 3.0,
        4.0, 5.0, 6.0,
        7.0, 8.0, 10.0
    };
    float B[n] = {1.0, 2.0, 3.0}; // The vector b

    // Device memory
    float *d_A = NULL;
    float *d_B = NULL;
    int *devIpiv = NULL, *devInfo = NULL;
    int lwork = 0;
    float *d_work = NULL;

    // Step 1: Create cuSolver handle, bind a stream
    hipsolverDnCreate(&cusolverH);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsolverSetStream(cusolverH, stream);

    // Step 2: Allocate memory on the device
    hipMalloc((void **)&d_A, sizeof(float) * lda * n);
    hipMalloc((void **)&d_B, sizeof(float) * n);
    hipMalloc((void **)&devIpiv, sizeof(int) * n);
    hipMalloc((void **)&devInfo, sizeof(int));

    // Copy matrices from host to device
    hipMemcpy(d_A, A, sizeof(float) * lda * n, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(float) * n, hipMemcpyHostToDevice);

    // Step 3: Query working space of getrf and getrs
    hipsolverDnSgetrf_bufferSize(cusolverH, n, n, d_A, lda, &lwork);
    hipMalloc((void **)&d_work, sizeof(float) * lwork);

    // Step 4: LU factorization
    hipsolverDnSgetrf(cusolverH, n, n, d_A, lda, d_work, devIpiv, devInfo);

    // Step 5: Solve Ax = b
    hipsolverDnSgetrs(cusolverH, HIPBLAS_OP_N, n, 1, d_A, lda, devIpiv, d_B, n, devInfo);

    // Copy result back to host
    hipMemcpy(B, d_B, sizeof(float) * n, hipMemcpyDeviceToHost);

    // Step 6: Check result
    printf("Solution: \n");
    for (int i = 0; i < n; i++) {
        printf("%f\n", B[i]);
    }

    // Clean up
    if (d_A) hipFree(d_A);
    if (d_B) hipFree(d_B);
    if (devIpiv) hipFree(devIpiv);
    if (devInfo) hipFree(devInfo);
    if (d_work) hipFree(d_work);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream) hipStreamDestroy(stream);

    printf("Completed Successfully!");

    return 0;
}
