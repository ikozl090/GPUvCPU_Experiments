#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <sys/time.h>
#include <lapacke.h>

int main(int argc, char *argv[]) {

    // Ensure matrix dimension was given
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <input_number> (Please enter matrix dimensions n for n x n matrix)\n", argv[0]);
        return 1;
    }

    int input = atoi(argv[1]);  // Convert the argument to an integer
    
    int MAX_PRINTABLE_MATRIX_DIM = 15; 

    // Print matrices if below max printable dimension 
    bool print_matrices = false;
    if (input <= MAX_PRINTABLE_MATRIX_DIM) {
        print_matrices = true; 
    }

    // Linear system size parameters 
    int n = input; // Matrix A is n x n and vector b is n x 1
    int rows_A = n; 
    int cols_A = n;
    int lda = rows_A; // leading dimension of array
    int ldb = rows_A; // leading dimension of array
    int nrhs = 1; // Number of right-hand sides (i.e., number of b vectors)

    // Allocate memory for matrix and vector 
    float *A, *d_A, *b, *d_b; 
    int size_A = sizeof(float) * rows_A * cols_A; 
    int size_b = sizeof(float) * lda; 
    A = (float *)malloc(size_A); 
    b = (float *)malloc(size_b); 

    // Initialize matrix and vector 
    double max_matrix_val = 1000;
    double min_matrix_val = -max_matrix_val;
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            A[n * i + j] = (double) rand() / ((double) RAND_MAX + 1) * (max_matrix_val - min_matrix_val) + min_matrix_val; 
        }
        b[i] = (double) rand() / ((double) RAND_MAX + 1) * (max_matrix_val - min_matrix_val) + min_matrix_val; 
    }

    // Print initial matrices if desirable
    if (print_matrices){
        // Print initialized matrix 
        printf("A = \n");
        for (int i = 0; i < rows_A; i++){
            for (int j = 0; j < cols_A; j++){
                printf(" %f ", A[rows_A * i + j]);
            }
            printf("\n");
        } 

        // Print initialized vector 
        printf("b = \n");
        for (int i = 0; i < rows_A; i++){
            printf(" %f ", b[i]);
            printf("\n");
        } 
    }

    /*********************************
        Perform Operations on GPU  
    **********************************/

    // Initialize start time variables 
    struct timeval start_time, end_time; 
    double run_time;
    gettimeofday(&start_time, NULL); // Get start time 

    // Variables to keep track of memory usage 
    size_t freeMemBefore, totalMemBefore, freeMemAfter, totalMemAfter;
    hipMemGetInfo(&freeMemBefore, &totalMemBefore); // Save initial memory before program exacution 

    // Allocate GPU memory for matrices 
    hipMalloc((void **)&d_A, size_A); 
    hipMalloc((void **)&d_b, size_b); 

    // Display initial and free memory 
    printf("Amout of free memory in GPU before exacution is %.4f GB out of %.4f GB total.\n", ((double)freeMemBefore)/(1000000000), ((double)totalMemBefore)/(1000000000));

    // Initialize and create cuSolver handler 
    hipsolverHandle_t solver_handle; 
    hipsolverDnCreate(&solver_handle); 

    // Transfer data from host to device (GPU) 
    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice); 
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice); 

    // Initialize variable for operation and buffer info
    int *devIpiv, *devInfo; // pivot and status information pointers 
    float *Workspace; // Buffer used during LU decomposition  
    int lwork = 0; // Size of workspace buffer used for operation 

    // Compute buffer size required for LU decomposition 
    hipsolverDnSgetrf_bufferSize(solver_handle, rows_A, cols_A, d_A, lda, &lwork);
    hipMalloc((void **)&devInfo, sizeof(int)); 
    hipMalloc((void **)&devIpiv, sizeof(int) * rows_A); 
    hipMalloc((void **)&Workspace, sizeof(float) *lwork);

    // Decompose system into LU matrices 
    hipsolverDnSgetrf(solver_handle, rows_A, cols_A, d_A, lda, Workspace, devIpiv, devInfo); 

    // Solve system in LU decomposed form 
    hipsolverDnSgetrs(solver_handle, HIPBLAS_OP_N, rows_A, cols_A, d_A, lda, devIpiv, d_b, lda, devInfo); 
    
    // Check devInfo to ensure cuSOLVER routine went well 
    int devInfo_h = 0; // dev info hat 
    hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost); // Save devInfo from GPU to devInfo_h on CPU
    if (devInfo_h != 0) {
        fprintf(stderr, "LU decomposition failed\n"); 
    }

    // Copy results to CPU 
    float* x = (float*)malloc(rows_A * sizeof(float)); 
    hipMemcpy(x, d_b, sizeof(float) * rows_A, hipMemcpyDeviceToHost); 

    // Get memory after solver execution 
    hipMemGetInfo(&freeMemAfter, &totalMemAfter);

    // Print results
    if (print_matrices){
        printf("x = \n");
        for (int idx = 0; idx < ldb; idx++) {
            printf(" %f ", x[idx]);
            printf("\n");
        }
        printf("\n");
    }

    // Print memory usage results 
    printf("Memory used by cuSOLVER function: %.4f GB out of %.4f GB total.\n", ((double)(freeMemBefore - freeMemAfter))/1000000000, ((double)totalMemBefore)/1000000000);

    // Free up memory 
    hipFree(Workspace);
    hipFree(devIpiv); 
    hipFree(devInfo); 
    hipFree(d_A); 
    hipFree(d_b); 
    free(x); 

    // Get end time 
    gettimeofday(&end_time, NULL);

    // Print running time 
    run_time = (double) (end_time.tv_sec - start_time.tv_sec); 
    run_time += (double) (end_time.tv_usec - start_time.tv_usec)/1000000; 
    printf("Total run time: %.4f seconds. \n", run_time);

    printf("Completed Successfully!\n");

    /*********************************
        Perform Operations on CPU  
    **********************************/

    // Initialize start time variables 
    struct timeval start_time_cpu, end_time_cpu; 
    double run_time_cpu;
    gettimeofday(&start_time_cpu, NULL); // Get start time 

    // Initialize variable for matrix operations 
    int ipiv[n]; // Variable for keeping track of pivot indices 
    int info; // To keep track of operation status 
    

    // Perform LU decomposition of A
    setenv("OPENBLAS_NUM_THREADS", "1", 1); // Only use one thread for CPU computation 
    info = LAPACKE_sgetrf(LAPACK_ROW_MAJOR, n, n, A, lda, ipiv);

    if (info > 0) {
        printf("The factorization has a zero diagonal element %d.\n", info);
        return -1;
    }

    // Solve the system Ax = b
    setenv("OPENBLAS_NUM_THREADS", "1", 1); // Only use one thread for CPU computation 
    info = LAPACKE_sgetrs(LAPACK_ROW_MAJOR, 'N', n, nrhs, A, lda, ipiv, b, ldb);

    if (info > 0) {
        printf("The solve operation failed %d.\n", info);
        return -1;
    }

    // Print results
    if (print_matrices){
        printf("x = \n");
        for (int idx = 0; idx < ldb; idx++) {
            printf(" %f ", b[idx]);
            printf("\n");
        }
        printf("\n");
    }

    // Get end time 
    gettimeofday(&end_time_cpu, NULL);

    // Print running time 
    run_time_cpu = (double) (end_time_cpu.tv_sec - start_time_cpu.tv_sec); 
    run_time_cpu += (double) (end_time_cpu.tv_usec - start_time_cpu.tv_usec)/1000000; 
    printf("Total run time (CPU): %.4f seconds. \n", run_time_cpu);

    return 0;
}
